#include "hip/hip_runtime.h"
/* File:     matrix vector multiplication cuda
 * Purpose:  Implement  on a gpu using cuda
 *
 */

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hipblas.h>

#include "../../util/common.h"

#define TOLERANCE 200.0f

using namespace std;

vector<data_t> A;
vector<data_t> B;
vector<data_t> C;

// Params ---------------------------------------------------------------------
typedef struct Params
{
    uint64_t row, column;
    bool shouldVerify;
} Params;

void usage()
{
    fprintf(stderr,
            "\nUsage:  ./gemv [options]"
            "\n"
            "\n    -r <R>    row size"
            "\n    -c <C>    column size"
            "\n    -v    t = verifies PIM output with host output. (default=false)"
            "\n");
}

struct Params input_params(int argc, char **argv)
{
    struct Params p;
    p.row = 65536;
    p.column = 65536;

    int opt;
    while ((opt = getopt(argc, argv, ":r:c:h:v:")) >= 0)
    {
        switch (opt)
        {
        case 'h':
            usage();
            exit(0);
            break;
        case 'r':
            p.row = atoll(optarg);
            break;
        case 'c':
            p.column = atoll(optarg);
            break;
        case 'v':
            p.shouldVerify = (*optarg == 't') ? true : false;
            break;
        default:
            fprintf(stderr, "\nUnrecognized option!\n");
            usage();
            exit(0);
        }
    }

    return p;
}

int main(int argc, char *argv[])
{
    struct Params p = input_params(argc, argv);

    u64 row = p.row, col = p.column;
    initVector(row * col, A);
    initVector(col, B);
    C.resize(row);

    float *x, *y, *z;

    hipError_t errorCode;

    errorCode = hipMalloc((void **)&x, row * col * sizeof(data_t));
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }
    errorCode = hipMalloc((void **)&y, col * sizeof(data_t));
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }
    errorCode = hipMalloc((void **)&z, row * sizeof(data_t));
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }

    errorCode = hipMemcpy(x, A.data(), row * col * sizeof(float), hipMemcpyHostToDevice);
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }

    errorCode = hipMemcpy(y, B.data(), col * sizeof(float), hipMemcpyHostToDevice);
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }

    const float alpha = 1.0;
    const float beta = 0.0;
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "CUBLAS initialization failed\n";
        exit(1);
    }

    // Event creation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float timeElapsed = 0;

    // Start timer
    hipEventRecord(start, 0);
    /* Kernel Call */
    status = hipblasSgemv(handle, HIPBLAS_OP_N, row, col, &alpha, x, row, y, 1, &beta, z, 1);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "CUBLAS SGEMV failed\n";
        exit(1);
    }

    // Check for kernel launch errors
    errorCode = hipGetLastError();
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }

    // End timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeElapsed, start, stop);

    printf("Execution time = %f ms\n", timeElapsed);

    errorCode = hipMemcpy(C.data(), z, row * sizeof(data_t), hipMemcpyDeviceToHost);
    if (errorCode != hipSuccess)
    {
        cerr << "Cuda Error Copy from host to device: " << hipGetErrorString(errorCode) << "\n";
        exit(1);
    }

    if (p.shouldVerify)
    {
        cout.precision(0);
        for (int i = 0; i < row; ++i)
        {
            data_t sum = 0;
            for (int j = 0; j < col; ++j)
            {
                sum += A[i + j * row] * B[j];
            }
            if (abs(C[i] - sum) > TOLERANCE)
            {
                cout << fixed << "Multiplication failed at index: " << i << "\t" << C[i] << "\t" << sum << endl;
                break;
            }
        }
        cout << "All correct!" << endl;
    }

    /* Free memory */
    hipblasDestroy(handle);
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
} /* main */
